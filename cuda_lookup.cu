#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32  // 定义每个块的大小

__global__ void lookupCUDA(float *output, float *source, int *index, int L, int K) {
    int row = blockIdx.x;
    int group = blockIdx.y;
    int tid = threadIdx.x;  // idx in index
    int wrap_start_idx = blockIdx.y * blockDim.x;
    int wrap_end_idx = (blockIdx.y + 1) * blockDim.x;
    if (wrap_start_idx <= index[tid] && index[tid] < wrap_end_idx) {
        output[blockIdx.x * blockDim.x + tid] = source[blockIdx.x * K + index[tid]];
    }
}

int main() {
    for (int L = 64; L <= 1024; L += 64) {
        for (int x = 1; x <= 4; x++) {
            int K = BLOCK_SIZE * x;    // source的行数，32的倍数
            int M = BLOCK_SIZE;    // index的大小

            // 分配主机内存
            float *h_source = (float *)malloc(K * L * sizeof(float));
            int *h_index = (int *)malloc(M * sizeof(int));
            float *h_output = (float *)malloc(M * L * sizeof(float));

            // 初始化source和index
            for (int i = 0; i < K * L; ++i) {
                h_source[i] = (float)i;
            }
            for (int i = 0; i < M; ++i) {
                h_index[i] = i * 2;  // 假设简单地查找前M个元素
            }
            // for (int i = 0; i < L; ++i) {
            //     for (int j = 0; j < K; ++j) {
            //         printf("%f ", h_source[i * K + j]);
            //     }
            //     printf("\n");
            // }
            //     printf("\n");
            // for (int i = 0; i < L; ++i) {
            //     for (int j = 0; j < M; ++j) {
            //         printf("%f ", h_source[i * K + h_index[j]]);
            //     }
            //     printf("\n");
            // }

            // 分配设备内存
            float *d_source, *d_output;
            int *d_index;
            hipMalloc((void **)&d_source, L * K * sizeof(float));
            hipMalloc((void **)&d_index, M * sizeof(int));
            hipMalloc((void **)&d_output, L * M * sizeof(float));

            // 将数据从主机复制到设备
            hipMemcpy(d_source, h_source, K * L * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_index, h_index, M * sizeof(int), hipMemcpyHostToDevice);

            // 定义块的大小
            dim3 numBlocks(L, K / BLOCK_SIZE);
            dim3 threadsPerBlock(BLOCK_SIZE);

            // 启动计时
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);


            // 启动核函数
            lookupCUDA<<<numBlocks, threadsPerBlock>>>(d_output, d_source, d_index, L, K);

            // 停止计时
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("[L=%d, K=%d, M=%d] Kernel execution time: %f ms\n", L, K, M, milliseconds);

            // 将结果从设备复制回主机
            hipMemcpy(h_output, d_output, M * L * sizeof(float), hipMemcpyDeviceToHost);

            // 验证结果
            bool correct = true;
            for (int i = 0; i < L; ++i) {
                for (int j = 0; j < M; ++j) {  // h_source[i * K + h_index[j]]
                    // printf("(%f %f)", h_output[i * M + j], h_source[i * K + h_index[j]]);
                    if (h_output[i * M + j] != h_source[i * K + h_index[j]]) {
                        correct = false;
                    }
                }
                // printf("\n");
            }

            if (correct) {
                printf("Lookup is correct.\n");
            } else {
                printf("Lookup is incorrect.\n");
            }

            // 释放内存
            free(h_source);
            free(h_index);
            free(h_output);
            hipFree(d_source);
            hipFree(d_index);
            hipFree(d_output);
            hipEventDestroy(start);
            hipEventDestroy(stop);
        }
    }


    

    return 0;
}
