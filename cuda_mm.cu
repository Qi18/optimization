#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16  // 定义每个块的大小

__global__ void matrixMulCUDA(float *C, float *A, float *B, int M, int K, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    for(int M = 64; M <= 1024; M += 64) {
        for(int N = 64; N <= 1024; N += 64) {
            for(int K = 64; K <= 1024; K += 64) {
                // 分配主机内存
                float *h_A = (float *)malloc(M * K * sizeof(float));
                float *h_B = (float *)malloc(K * N * sizeof(float));
                float *h_C = (float *)malloc(M * N * sizeof(float));

                // 初始化矩阵A和矩阵B
                for (int i = 0; i < M * K; ++i) {
                    h_A[i] = 1.0f;
                }
                for (int i = 0; i < K * N; ++i) {
                    h_B[i] = 2.0f;
                }

                // 分配设备内存
                float *d_A, *d_B, *d_C;
                hipMalloc((void **)&d_A, M * K * sizeof(float));
                hipMalloc((void **)&d_B, K * N * sizeof(float));
                hipMalloc((void **)&d_C, M * N * sizeof(float));

                // 将矩阵A和矩阵B从主机复制到设备
                hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

                // 定义网格和块的大小
                dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
                dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

                // 启动计时
                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);
                hipEventRecord(start);
                // 启动核函数
                matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, M, K, N);

                hipEventRecord(stop);
                hipEventSynchronize(stop);

                float milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                printf("[M=%d, K=%d, N=%d] Kernel execution time: %f ms\n", M, K, N, milliseconds);

                // 将结果从设备复制回主机
                hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

                // 验证结果
                bool correct = true;
                for (int i = 0; i < M * N; ++i) {
                    if (h_C[i] != K * 2) {
                        correct = false;
                        break;
                    }
                }

                if (correct) {
                    printf("Matrix multiplication is correct.\n");
                } else {
                    printf("Matrix multiplication is incorrect.\n");
                }

                // 释放内存
                free(h_A);
                free(h_B);
                free(h_C);
                hipFree(d_A);
                hipFree(d_B);
                hipFree(d_C);

            }
        }

    }

    return 0;
}
